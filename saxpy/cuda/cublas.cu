#include <hipblas.h>
#include <stdlib.h>
#include <stdio.h>

#include "timer.h"

__host__ void saxpy(int _N) {
    int N = _N * _N; // mimic a 2-D array
    
    // Handlers
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipError_t    err;

    stat = hipblasCreate(&handle);

    float* x = nullptr;
    float* y = nullptr;
    float* z = nullptr;

    float* d_x = nullptr;
    float* d_y = nullptr;
    float* d_z = nullptr;

    // Memory allocations
    err = hipHostMalloc(&x, 3 * N * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate memory on host.\n");
        exit(-1);
    }
    y = x + N;
    z = y + N;
    err = hipMalloc(&d_x, 3 * N * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate memory on device. Requested size %lu\n", 3 * N * sizeof(float));
        exit(-1);
    }
    d_y = d_x + N;
    d_z = d_y + N;

    for(int i = 0; i < 3 * N; ++i) {
        x[i] = rand() / RAND_MAX;
    }

    // Memory set on GPU
    hipblasSetVector(N, sizeof(float), x, 1, d_x, 1);
    hipblasSetVector(N, sizeof(float), y, 1, d_y, 1);
    float alpha = 2.0;
    float alpha_1 = 4.0;
    
    // Bechmark loop
    int nIter = 5000;
    Timer tmr;
    tmr.start();
    for (int i = 0; i < nIter; ++i) {
        hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1);
        hipblasSaxpy(handle, N, &alpha_1, d_x, 1, d_y, 1);
    }
    hipblasSetVector(N, sizeof(float), d_z, 1, z, 1);
    tmr.stop();

    // Performance report
    double avg_time = tmr.getTimeMillisecond() / nIter;
    double GFlops = 1e-6 * N * 2 * 2/ avg_time;
    double GBs = 1e-6 * N * sizeof(float) * 3 / avg_time;
    printf("%dx%d, time %.3lf ms, %.3lf GFLOPS, %.3lf GB/s\n", _N, _N, avg_time, GFlops, GBs);

    // Clean up
    hipblasDestroy(handle);
    hipFree(x);
    hipFree(d_x);
    hipFree(y);
    hipFree(d_y);
    hipFree(z);
    hipFree(d_z);
}

int main() {
    int N = 256;
    for(int i = 0; i < 5; ++i) {
        saxpy(N);
        N *= 2;
    }
    return 0;
}
