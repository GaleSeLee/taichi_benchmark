#include "scan.cuh"

int main(int argc, char **argv) {
    int num_items = 4096;
    if(argc > 1) num_items = std::atoi(argv[1]);
    float *h_in = new float [num_items];
    float *h_reference = new float[num_items];
    float *h_out = new float [num_items + 1];
    thrust::device_vector<float> u(num_items);
    Initialize(h_in, num_items);
    Solve(h_in, h_reference, num_items);

    cuErrCheck(hipMemcpy(thrust::raw_pointer_cast(&u[0]), h_in, sizeof(float) * num_items, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    thrust::inclusive_scan(u.begin(), u.end(), u.begin());
    hipDeviceSynchronize();
    cuErrCheck(hipGetLastError());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f", milliseconds);
    cuErrCheck(hipMemcpy(h_out, thrust::raw_pointer_cast(&u[0]), sizeof(float) * (num_items), hipMemcpyDeviceToHost));
    TestResult(h_out, h_reference, num_items);

    delete[] h_in;
    delete[] h_out;
    delete[] h_reference;
}
