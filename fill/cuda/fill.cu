#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//#define USE_MEMSET

__global__ void FillByKernel(float *arr, float c, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    arr[i] = c;
  }
}

int main(int argc, char** argv)
{
  int numElements = std::atoi(argv[1]);
#ifdef USE_MEMSET
  CUdeviceptr d_buf;
#else
  float *d_buf = NULL;
#endif

  float *h_out = new float [numElements];
 
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

#ifdef USE_MEMSET
  cuMemAlloc(&d_buf, numElements*sizeof(float));
#else
  hipMalloc(&d_buf, numElements*sizeof(float));
#endif

  int threadsPerBlock = 128;
  int blocksPerGrid = std::ceil(double(numElements)/double(threadsPerBlock));

  // dry run
  float v = 0.5;
#ifdef USE_MEMSET
  cuMemsetD32(d_buf, reinterpret_cast<uint32_t &>(v), numElements);
#else
  FillByKernel<<<blocksPerGrid, threadsPerBlock>>>(d_buf, 0.5, numElements);
#endif

  // measure
  int num_runs = 500;
  hipEventRecord(start);
  for (int i=0; i<num_runs; i++) {
#ifdef USE_MEMSET
    cuMemsetD32(d_buf, reinterpret_cast<uint32_t &>(v), numElements);
#else
    FillByKernel<<<blocksPerGrid, threadsPerBlock>>>(d_buf, 0.5, numElements);
#endif
  }
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("fill %i elements takes %f ms\n", numElements, milliseconds/num_runs);
  
#ifdef USE_MEMSET
  cuMemFree(d_buf);
#else
  hipFree(d_buf);
#endif
  delete[] h_out;
  return 0;
}
